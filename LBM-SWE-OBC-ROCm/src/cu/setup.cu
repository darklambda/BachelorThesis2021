#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "../include/structs.h"

__global__ void auxArraysKernel(int Lx, int Ly,
	const int* __restrict__ ex, const int* __restrict__ ey,
	const int* __restrict__ node_types,
	unsigned char* SC_bin, unsigned char* BB_bin) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int size = Lx * Ly;
	if (i < size) {
		int y = (int)i / Lx;
		int x = i - y * Lx;
		int xi, yi, ind, indj, indk, a;
		int valueSC = 0, valueBB = 0;
		if (node_types[i] == 2) {
			if (y == 0) {
				if (x == 0) 
					valueSC += 4 + 8 + 64;
				else if (x == Lx - 1)
					valueSC += 1 + 8 + 128;
				else 
					valueSC += 1 + 4 + 8 + 64 + 128;
			}
			else if (y == Ly - 1) {
				if (x == 0) 
					valueSC += 2 + 4 + 32;
				else if (x == Lx - 1) 
					valueSC += 1 + 2 + 16;
				else 
					valueSC += 1 + 2 + 4 + 16 + 32;
			}
			else {
				if (x == 0)
					valueSC += 2 + 4 + 8 + 32 + 64;
				else if (x == Lx - 1) 
					valueSC += 1 + 2 + 8 + 16 + 128;
				else  
					valueSC = 255;
			}
		}
		else if (node_types[i] == 1) {
			if (y == 0) {
				valueSC += 1 + 8 + 128;
				valueBB += 4 + 32 + 64;
			}
			else if (y == Ly - 1) {
				valueSC += 1 + 2 + 16;
				valueBB += 4 + 32 + 64;
			}
			else {
				for (a = 1; a<9; a++) {
					yi = y - ey[a];
					xi = x - ex[a];
					ind = yi * Lx + xi;
					if (node_types[ind] != 0) 
						valueSC += (1 << (a-1));
					else 
						valueBB += (1 << (a-1));
					if (a > 4) {
						if (node_types[ind] == 1) {
							indj = y * Lx + xi;
							indk = yi * Lx + x;
							if (node_types[indj] == 0 || node_types[indk] == 0) {
								valueSC -= (1 << (a-1));
								valueBB += (1 << (a-1));
							}
						}
					}
				}
			}
		}
		SC_bin[i] = (unsigned char) valueSC;
		BB_bin[i] = (unsigned char) valueBB;
	}
} 

__global__ void hKernel(int Lx, int Ly, const prec* __restrict__ w,
	const prec* __restrict__ b, prec* h) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < Lx*Ly) {
		h[i] = w[i] - b[i];
	}
}
