#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "include/LBMkernels.cuh"
#include "include/utils.cuh"
#include "include/SWE.cuh"
#include "include/PDEfeq.cuh"
#include "include/BC.cuh"
#include "../include/structs.h"
#include "../include/macros.h"
 
__device__ void calculateMacroscopic(prec* localMacroscopic, prec* localf, prec e, int i){
	localMacroscopic[3*i] = localf[9*i] + (localf[9*i+1] + localf[9*i+2] + localf[9*i+3] + localf[9*i+4]) + (localf[9*i+5] + localf[9*i+6] + localf[9*i+7] + localf[9*i+8]);
	localMacroscopic[3*i+1] = e * ((localf[9*i+1] - localf[9*i+3]) + (localf[9*i+5] - localf[9*i+6] - localf[9*i+7] + localf[9*i+8])) / localMacroscopic[3*i];
	localMacroscopic[3*i+2] = e * ((localf[9*i+2] - localf[9*i+4]) + (localf[9*i+5] + localf[9*i+6] - localf[9*i+7] - localf[9*i+8])) / localMacroscopic[3*i];
}

__global__ void First(const configStruct config, prec* localMacroscopic, prec* forcing, prec* localf, 
	const prec* __restrict__ b, const unsigned char* __restrict__ binary1, 
	const unsigned char* __restrict__ binary2, const prec* __restrict__ f1, 
	prec* f2, prec* h) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;	
	if (i < config.Lx*config.Ly) {
		unsigned char b1 = binary1[i];
		unsigned char b2 = binary2[i];
		if(b1 != 0 || b2 != 0){
			int ex[8] = {1,0,-1,0,1,-1,-1,1};		
			int ey[8] = {0,1,0,-1,1,1,-1,-1};
			#if PDE == 1
				prec factor = 1 / (6 * config.e*config.e);
				prec localh = h[i];
				prec localb = b[i];
				for (int j = 0; j < 4; j++){
					int index = IDX(i, j, config.Lx, ex, ey);
					if (index > 0 && index < config.Lx*config.Ly) {
					forcing[8*i+j] = factor * 9.8 * (localh + h[index]) * (b[index] - localb);
					} else {
						forcing[8*i+j] = 0.0;
					}
				}
				for (int j = 4; j < 8; j++){
					int index = IDX(i, j, config.Lx, ex, ey);
					if (index > 0 && index < config.Lx*config.Ly) {
					forcing[8*i+j] = factor * 0.25 * 9.8 * (localh + h[index]) * (b[index] - localb);
					} else {
						forcing[8*i+j] = 0.0;
					}
				}
			#elif PDE == 5
				calculateForcingUser(forcing, h, b, config.e, i, config.Lx, ex, ey);
			#else 
				for (int j = 0; j < 8; j++)
					forcing[8*i+j] = 0;
			#endif


			localf[9*i] = f1[9*i]; 
			for (int j = 1; j < 9; j++){
				if(((b1>>(j-1)) & 1) & (~(b2>>(j-1)) & 1)) 
					localf[9*i+j] = f1[IDXcm(IDX(i, j-1, config.Lx, ex, ey), j, config.Lx, config.Ly)] + forcing[8*i+j-1];
				else if((~(b1>>(j-1)) & 1) & (~(b2>>(j-1)) & 1)) 
					localf[9*i+j] = f1[IDXcm(i, j, config.Lx, config.Ly)];
			}

			for (int j = 1; j < 9; j++)
				if((~(b1>>(j-1)) & 1) & ((b2>>(j-1)) & 1)) 
					#if BC1 == 1
						OBC(localf, f1, i, j, config.Lx, config.Ly);
					#elif BC1 == 2
						PBC(localf, f1, i, j, config.Lx, config.Ly, ex, ey);
					#elif BC1 == 3
						BBBC(localf, j);
					#elif BC1 == 4
						SBC(localf, j, b1, b2);
					#elif BC1 == 5
						UBC1(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#elif BC1 == 6
						UBC2(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#endif

			#if BC2 != 0
			for (int j = 1; j < 9; j++)
				if(((b1>>(j-1)) & 1) & ((b2>>(j-1)) & 1)) 
					#if BC2 == 1
						localf[9*i+j] = OBC(localf, f1, i, j, config.Lx, config.Ly);
					#elif BC2 == 2
						localf[9*i+j] = PBC(localf, f1, i, j, config.Lx, config.Ly, ex, ey);
					#elif BC2 == 3
						localf[9*i+j] = BBBC(localf, j);
					#elif BC2 == 4
						localf[9*i+j] = SBC(localf, j, b1, b2);
					#elif BC2 == 5
						localf[9*i+j] = BC1User(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#elif BC2 == 6
						localf[9*i+j] = BC2User(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#endif
			#endif
			
		}
	} 
} 

__global__ void Second(const configStruct config, prec* localMacroscopic, prec* forcing, prec* localf, 
	const prec* __restrict__ b, const unsigned char* __restrict__ binary1, 
	const unsigned char* __restrict__ binary2, const prec* __restrict__ f1, 
	prec* f2, prec* h) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;	
	if (i < config.Lx*config.Ly) {
		unsigned char b1 = binary1[i];
		unsigned char b2 = binary2[i];
		if(b1 != 0 || b2 != 0){
			calculateMacroscopic(localMacroscopic, localf, config.e, i);
			h[i] = (prec)localMacroscopic[3*i];

		}
	}
}

__global__ void Third(const configStruct config, prec* localMacroscopic, prec* forcing, prec* localf, 
	const prec* __restrict__ b, const unsigned char* __restrict__ binary1, 
	const unsigned char* __restrict__ binary2, const prec* __restrict__ f1, 
	prec* f2, prec* h) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;	
	if (i < config.Lx*config.Ly) {
		unsigned char b1 = binary1[i];
		unsigned char b2 = binary2[i];
		if(b1 != 0 || b2 != 0){
			prec localMacroscopicTmp[3];

			localMacroscopicTmp[0] = (prec)localMacroscopic[3*i];
			localMacroscopicTmp[1] = (prec)localMacroscopic[3*i+1];
			localMacroscopicTmp[2] = (prec)localMacroscopic[3*i+2];

			prec feq[9];
			#if PDE == 1
				calculateFeqSWE(feq, localMacroscopicTmp, config.e);
			#elif PDE == 2
				calculateFeqHE(feq, localMacroscopicTmp, config.e);
			#elif PDE == 3
				calculateFeqWE(feq, localMacroscopicTmp, config.e);
			#elif PDE == 4
				calculateFeqNSE(feq, localMacroscopicTmp, config.e);
			#elif PDE == 5
				calculateFeqUser(feq, localMacroscopicTmp, config.e);
			#endif
			
			for (int j = 0; j < 9; j++)
				f2[IDXcm(i, j, config.Lx, config.Ly)] = localf[9*i+j] - (localf[9*i+j] - feq[j]) / config.tau;
		}
	}
}